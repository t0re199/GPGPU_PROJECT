#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdlib.h>
#include <assert.h>
#include <stdio.h>

#define THREADS 512
#define BLOCKS 32
#define TILE_SIZE 1024

#define min(v1, v2) v1 < v2 ? v1 : v2

void assertfy(const int *buffer, const int len);
void printUpTo(int *buffer, const int len);

void assertfy(const int *buffer, const int len)
{
    int i = 1;

    for (; i < len; i++)
    {
        assert(buffer[i] >= buffer[i - 1]);
    }
}

void printUpTo(int *buffer, const int len)
{
    int i = 0;
    for (; i < len; i++)
    {
        printf("%d\t", buffer[i]);
    }
    printf("\n");
}

__global__ void fillAscending(int *buffer, const int len)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len)
    {
        buffer[i] = i;
    }
}

__host__ __device__ 
int coRank(int k, int *A, int m, int *B, int n)
{
    int i = (k < m) ? k : m;
    int j = k - i;
    int i_low = (0 > (k - n)) ? 0 : k - n;
    int j_low = (0 > (k - m)) ? 0 : k - m;
    int delta;
    bool active = true;

    while (active)
    {
        if (i > 0 && j < n && A[i - 1] > B[j])
        {
            delta = ((i - i_low + 1) >> 1);
            j_low = j;
            j = j + delta;
            i = i - delta;
        }
        else if (j > 0 && i < m && B[j - 1] >= A[i])
        {
            delta = ((j - j_low + 1) >> 1);
            i_low = i;
            i = i + delta;
            j = j - delta;
        }
        else
        {
            active = false;
        }
    }

    return i;
}

__host__ __device__ void merge(const int *buff1, int m, const int *buff2, int n, int *acc)
{
    int i = 0x0,
        j = 0x0,
        k = 0x0;

    while ((i < m) && (j < n))
    {
        if (buff1[i] <= buff2[j])
        {
            acc[k++] = buff1[i++];
        }
        else
        {
            acc[k++] = buff2[j++];
        }
    }

    for (; i < m; i++)
    {
        acc[k++] = buff1[i];
    }

    for (; j < n; j++)
    {
        acc[k++] = buff2[j];
    }
}

__global__ 
void tiledMergeKernel(int *A, int m, int *B, int n, int *C, int tile_size)
{
    extern __shared__ int shareAB[];
    int *A_S = shareAB;
    int *B_S = shareAB + tile_size;
    int C_curr = blockIdx.x * ceil((m + n) / (float)gridDim.x);
    int C_next = min((blockIdx.x + 1) * (int)ceil((m + n) / (float)gridDim.x), m + n);

    if (threadIdx.x == 0)
    {
        A_S[0] = coRank(C_curr, A, m, B, n);
        A_S[1] = coRank(C_next, A, m, B, n);
    }
    __syncthreads();

    int A_curr = A_S[0];
    int A_next = A_S[1];
    int B_curr = C_curr - A_curr;
    int B_next = C_next - A_next;
    __syncthreads();

    int counter = 0;
    int C_length = C_next - C_curr;
    int A_length = A_next - A_curr;
    int B_length = B_next - B_curr;
    int total_iteration = ceil((C_length) / (float)tile_size);
    int C_completed = 0;
    int A_consumed = 0;
    int B_consumed = 0;

    while (counter < total_iteration)
    {
        for (int i = 0; i < tile_size; i += blockDim.x)
        {
            if (i + threadIdx.x < A_length - A_consumed)
                A_S[i + threadIdx.x] = A[A_curr + A_consumed + i + threadIdx.x];
        }
        for (int i = 0; i < tile_size; i += blockDim.x)
        {
            if (i + threadIdx.x < B_length - B_consumed)
                B_S[i + threadIdx.x] = B[B_curr + B_consumed + i + threadIdx.x];
        }
        __syncthreads();

        int c_curr = threadIdx.x * (tile_size / blockDim.x);
        int c_next = (threadIdx.x + 1) * (tile_size / blockDim.x);
        c_curr = (c_curr <= C_length - C_completed) ? c_curr : C_length - C_completed;
        c_next = (c_next <= C_length - C_completed) ? c_next : C_length - C_completed;

        int a_curr = coRank(c_curr, A_S, min(tile_size, A_length - A_consumed), 
                            B_S, min(tile_size, B_length - B_consumed));
        int b_curr = c_curr - a_curr;
        int a_next = coRank(c_next, A_S, min(tile_size, A_length - A_consumed), 
                            B_S, min(tile_size, B_length - B_consumed));
        int b_next = c_next - a_next;

        merge(A_S + a_curr, a_next - a_curr,
              B_S + b_curr, b_next - b_curr,
              C + C_curr + C_completed + c_curr);
              
        counter++;
        C_completed += tile_size;
        A_consumed += coRank(tile_size, A_S, tile_size, B_S, tile_size);
        B_consumed = C_completed - A_consumed;
        __syncthreads();
    }
}

int main(int argc, char const *argv[])
{
    int m = 1 << 24,
        n = 1 << 24;

    int *A,
        *B,
        *C;

    hipMallocManaged((void **)&A, m * sizeof(int), hipMemAttachGlobal);
    hipMallocManaged((void **)&B, n * sizeof(int), hipMemAttachGlobal);

    hipMallocManaged((void **)&C, (n + m) * sizeof(int), hipMemAttachGlobal);

    fillAscending<<<ceil((double)m / THREADS), THREADS>>>(A, m);
    fillAscending<<<ceil((double)n / THREADS), THREADS>>>(B, n);

    tiledMergeKernel<<<BLOCKS, THREADS, TILE_SIZE * 2 * sizeof(int)>>>(A, m, B, n, C, TILE_SIZE);

    hipDeviceSynchronize();

    assertfy(C, m + n);
    printUpTo(C, 10);

    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}
